#include "hip/hip_runtime.h"
﻿
#include "../../../common/mlcudaCommon.h"
#include "mrConstantParamsGpu3D.h"
#include "mrUtilFuncGpu3D.h"
#include "mrLbmSolverGpu3D.h"
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/host_vector.h>
#include "tDCCL.cuh"


__host__ __device__
void MomSwap(REAL*& pt1, REAL*& pt2) {
	REAL* temp = pt1;
	pt1 = pt2;
	pt2 = temp;
}


__host__ __device__
void MomSwap(double*& pt1, double*& pt2) {
	double* temp = pt1;
	pt1 = pt2;
	pt2 = temp;
}

__host__ __device__ inline void swap(int& a, int& b) {
	int temp = a;
	a = b;
	b = temp;
}

__device__ inline void swap(float& a, float& b) {
	int temp = a;
	a = b;
	b = temp;
}

__device__ inline void swap(double& a, double& b) {
	int temp = a;
	a = b;
	b = temp;
}

__device__ inline void swap(float3& a, float3& b) {
	float3 temp = a;
	a = b;
	b = temp;
}


__device__ void report_split(mrFlow3D* mlflow, int x, int y, int z, int sample_x, int sample_y, int sample_z)
{
	int curind = z * sample_y * sample_x + y * sample_x + x;
	// use previous tag to record for the bubble volume change computation
	mlflow[0].previous_tag[curind] = mlflow[0].tag_matrix[curind];
	mlflow[0].tag_matrix[curind] = -1;
	if (mlflow[0].previous_tag[curind]>0)
		atomicExch(&mlflow[0].split_flag, 1);
}


__global__ void clear_detector(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		mlflow[0].merge_detector[curind] = 0;
	}
	if (curind == 1)
	{
		mlflow[0].split_flag = 0;
		mlflow[0].merge_flag = 0;
	}
}

void ClearDectector(mrFlow3D* mlflow, MLFluidParam3D* param)
{
	int sample_x = param->samples.x;
	int sample_y = param->samples.y;
	int sample_z = param->samples.z;
	int sample_num = sample_x * sample_y;
	int total_num = sample_num * sample_z;
	dim3 threads1(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
	dim3 grid1(
		ceil(REAL(sample_x) / threads1.x),
		ceil(REAL(sample_y) / threads1.y),
		ceil(REAL(sample_z) / threads1.z)
	);

	clear_detector << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}


__global__ void clear_inlet(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int total_num)
{

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		if (mlflow[0].islet[curind] == 1)
		{
			mlflow[0].islet[curind] = 0;
			mlflow[0].flag[curind] = TYPE_G;
			mlflow[0].phi[curind] = 0.f;
			mlflow[0].mass[curind] = 0.f; // update mass
			mlflow[0].massex[curind] = 0.f; // update mass
			mlflow[0].fMom[curind + 0 * total_num] = 1.f;
			mlflow[0].fMom[curind + 1 * total_num] = 0.f;
			mlflow[0].fMom[curind + 2 * total_num] = 0.f;
			mlflow[0].fMom[curind + 3 * total_num] = 0.f;
			mlflow[0].fMom[curind + 4 * total_num] = 0.f;
			mlflow[0].fMom[curind + 5 * total_num] = 0.f;
			mlflow[0].fMom[curind + 6 * total_num] = 0.f;
			mlflow[0].fMom[curind + 7 * total_num] = 0.f;
			mlflow[0].fMom[curind + 8 * total_num] = 0.f;
			mlflow[0].fMom[curind + 9 * total_num] = 0.f;	
		}
	}
}

__global__ void ResetDisjoinForce(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;

	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		mlflow[0].massex[curind] = 0.0f;
		mlflow[0].disjoin_force[curind] = 0.f;
	}
}

__global__ void calculate_disjoint(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int total_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	int curind = z * sample_x * sample_y + y * sample_x + x;
	mrUtilFuncGpu3D mrutilfunc;

	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		const unsigned char flagsn = mlflow[0].flag[curind]; // cache flags[n] for multiple readings
		const unsigned char flagsn_bo = flagsn & TYPE_BO, flagsn_su = flagsn & TYPE_SU; // extract boundary and surface flags
		if (flagsn_su == TYPE_I)
		{
			float massn = mlflow[0].mass[curind];
			float phij[27]; // cache fill level of neighbor lattice points
			for (int i = 1; i < 27; i++)
			{
				int dx = int(ex3d_gpu[i]);
				int dy = int(ey3d_gpu[i]);
				int dz = int(ez3d_gpu[i]);
				int x1 = x - dx;
				int y1 = y - dy;
				int z1 = z - dz;

				int ind_back = z1 * sample_x * sample_y + y1 * sample_x + x1;
				massn += mlflow[0].massex[ind_back]; // get mass to derive the current phij[0]
			}
			for (int i = 1; i < 27; i++)
			{
				int dx = int(ex3d_gpu[i]);
				int dy = int(ey3d_gpu[i]);
				int dz = int(ez3d_gpu[i]);

				int x1 = x - dx;
				int y1 = y - dy;
				int z1 = z - dz;

				int ind_back = z1 * sample_x * sample_y + y1 * sample_x + x1;
				phij[i] = mlflow[0].phi[ind_back]; 

				if ((mlflow[0].flag[ind_back] & TYPE_BO) == TYPE_S)
				{
					for (int ijk = 0; ijk < 6; ijk++)
					{
						int x2 = x1 - int(ex3d_gpu[ijk + 1]);
						int y2 = y1 - int(ey3d_gpu[ijk + 1]);
						int z2 = z1 - int(ez3d_gpu[ijk + 1]);
						if (x2 >= 0 && x2 < sample_x && y2 >= 0 && y2 < sample_y && z2 >= 0 && z2 < sample_z)
						{
							int ind_k = z2 * sample_y * sample_x + y2 * sample_x + x2;
							if ((mlflow[0].flag[ind_k] & TYPE_BO) != TYPE_S)
							{
								phij[i] = mlflow[0].phi[ind_k];
								break;
							}
						}
					}
				}
			}
			float rhon = 0.0f;
			rhon = mlflow[0].fMom[total_num * 0 + curind];
			phij[0] = mrutilfunc.calculate_phi(rhon, massn, flagsn); 
			int tag_curind = mlflow[0].tag_matrix[curind] - 1;
			float3 normal = mrutilfunc.calculate_normal(phij);
			float disjoint = 0.f;
			int max_ids = -1;
		
			for (int jk = 1; jk < 20; jk++)
			{
				// start a ray along the normal direction for max(jk) * 0.2 = 4.0 length and check the interface
				int x12 = round((float)x - (float)jk * 0.2f * normal.x);
				int y12 = round((float)y - (float)jk * 0.2f * normal.y);
				int z12 = round((float)z - (float)jk * 0.2f * normal.z);

				if (x12 >= 0 && x12 < sample_x && y12 >= 0 && y12 < sample_y && z12 >= 0 && z12 < sample_z)
				{
					int ind_back = z12 * sample_x * sample_y + y12 * sample_x + x12;
					if (mlflow[0].tag_matrix[ind_back] > 0)
					{
						int tag_neighbor = mlflow[0].tag_matrix[ind_back] - 1;
						// refine the distance (consider the center offset for the current node and the opacity for the neighbor node)
						if (tag_curind != tag_neighbor && mlflow[0].flag[ind_back] == TYPE_I)
						{
							float center_offset = mrutilfunc.plic_cube(phij[0], normal);
							float alpha = mlflow[0].phi[ind_back];

							float dis = abs((float)jk * 0.2f * normal.x) - (1 - alpha);
							float d = abs(dis / (normal.x + 1e-8)) - center_offset;

							if (disjoint < 1.f - d / 4.f)
							{
								disjoint = 1.f - d / 4.f;
								max_ids = tag_neighbor;
							}
						}
					}
				}
			}
			if (disjoint > 0)
			{
				atomicAdd(&mlflow[0].disjoin_force[curind], disjoint);
			}
		}
	}
}

__global__ void Init3D(
	mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num, int total_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_num + y * sample_x + x;
	mrUtilFuncGpu3D mrutilfunc;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{

		unsigned char flagsn = mlflow[0].flag[curind];
		const unsigned char flagsn_bo = flagsn & TYPE_BO; // extract boundary flags

		unsigned char flagsj[27]{};
		for (int i = 0; i < 27; i++)
		{
			int dx = int(ex3d_gpu[i]);
			int dy = int(ey3d_gpu[i]);
			int dz = int(ez3d_gpu[i]);
			int x1 = x - dx;
			int y1 = y - dy;
			int z1 = z - dz;

			if (x1 >= 0 && x1 < sample_x && y1 >= 0 && y1 < sample_y && z1 >= 0 && z1 < sample_z)
			{
				int ind_back = z1 * sample_num + y1 * sample_x + x1;
				flagsj[i] = mlflow[0].flag[ind_back];
			}
		}

		if (flagsn_bo == TYPE_S) { // cell is solid
			bool TYPE_ONLY_S = true; // has only solid neighbors
			for (int i = 1; i < 27; i++) TYPE_ONLY_S = TYPE_ONLY_S && (flagsj[i] & TYPE_BO) == TYPE_S;
			if (TYPE_ONLY_S) {
				mlflow[0].fMomPost[curind + 1 * total_num] = mlflow[0].fMom[curind + 1 * total_num] = 0.0f; // reset velocity for solid lattice points with only boundary neighbors
				mlflow[0].fMomPost[curind + 2 * total_num] = mlflow[0].fMom[curind + 2 * total_num] = 0.0f;
				mlflow[0].fMomPost[curind + 3 * total_num] = mlflow[0].fMom[curind + 3 * total_num] = 0.0f;
			}
		}
		if (flagsn_bo == TYPE_S) {
			mlflow[0].fMomPost[curind + 1 * total_num] = mlflow[0].fMom[curind + 1 * total_num] = 0.0f; // reset velocity for solid lattice points with only boundary neighbors
			mlflow[0].fMomPost[curind + 2 * total_num] = mlflow[0].fMom[curind + 2 * total_num] = 0.0f;
			mlflow[0].fMomPost[curind + 3 * total_num] = mlflow[0].fMom[curind + 3 * total_num] = 0.0f;
		}

		// calculate the equilibrium distribution function
		float feq[27]{}; 
		mrutilfunc.calculate_f_eq(mlflow[0].fMom[curind + 0 * total_num], 
			mlflow[0].fMom[curind + 1 * total_num], mlflow[0].fMom[curind + 2 * total_num], 
			mlflow[0].fMom[curind + 3 * total_num], feq);

		float geq[7]{};
		mrutilfunc.calculate_g_eq(mlflow[0].c_value[curind], 
			mlflow[0].fMom[curind + 1 * total_num], mlflow[0].fMom[curind + 2 * total_num], 
			mlflow[0].fMom[curind + 3 * total_num], geq);


		// reset the flag for the interface between fluid and gas
		float phin = mlflow[0].phi[curind];
		if (!(flagsn & (TYPE_S | TYPE_E | TYPE_T | TYPE_F | TYPE_I)))
			flagsn = (flagsn & ~TYPE_SU) | TYPE_G; // change all non-fluid and non-interface flags to gas
		if ((flagsn & TYPE_SU) == TYPE_G)
		{ // cell with updated flags is gas
			bool change = false; // check if cell has to be changed to interface
			for (int i = 1; i < 27; i++)
				change = change || (flagsj[i] & TYPE_SU) == TYPE_F; // if neighbor flag fluid is set, the cell must be interface
			if (change)
			{ // create interface automatically if phi has not explicitely defined for the interface layer
				flagsn = (flagsn & ~TYPE_SU) | TYPE_I; // cell must be interface
				phin = 0.5f;
				float rhon, uxn, uyn, uzn; // initialize interface cells with average density/velocity of fluid neighbors
				 // average over all fluid/interface neighbors
				float rhot = 0.0f, uxt = 0.0f, uyt = 0.0f, uzt = 0.0f, counter = 0.0f; // average over all fluid/interface neighbors
				float rhon_g = 0.0f ,rhogt = 0.0f, c_k = 0.0f; // average over all fluid/interface neighbors
				for (int i = 1; i < 27; i++)
				{
					int dx = int(ex3d_gpu[i]);
					int dy = int(ey3d_gpu[i]);
					int dz = int(ez3d_gpu[i]);
					int x1 = x - dx;
					int y1 = y - dy;
					int z1 = z - dz;
					int ind_back = z1 * sample_num + y1 * sample_x + x1;
					const unsigned char flagsji_su = mlflow[0].flag[ind_back] & TYPE_SU;
					if (flagsji_su == TYPE_F) { // fluid or interface or (interface->fluid) neighbor
						counter += 1.0f;
						rhot += mlflow[0].fMom[ind_back + 0 * total_num];
						uxt += mlflow[0].fMom[ind_back + 1 * total_num];
						uyt += mlflow[0].fMom[ind_back + 2 * total_num];
						uzt += mlflow[0].fMom[ind_back + 3 * total_num];
						if (i < 7)
						{
							c_k += 1.0f;
							rhogt += mlflow[0].c_value[ind_back];
						}
					}
				}
				rhon = counter > 0.0f ? rhot / counter : 1.0f;
				uxn = counter > 0.0f ? uxt / counter : 0.0f;
				uyn = counter > 0.0f ? uyt / counter : 0.0f;
				uzn = counter > 0.0f ? uzt / counter : 0.0f;
				rhon_g = c_k > 0.0f ? rhogt / c_k : 0.0f;

				mrutilfunc.calculate_f_eq(rhon, uxn, uyn, uzn, feq); // calculate equilibrium DDFs
				mrutilfunc.calculate_g_eq(rhon_g, uxn, uyn, uzn, geq);
				mlflow[0].c_value[curind] = rhon_g;
			}
		}
		if ((flagsn & TYPE_SU) == TYPE_G) { // cell with updated flags is still gas
			mlflow[0].fMom[curind + 1 * total_num] = 0.0f; // reset velocity for solid lattice points with only boundary neighbors
			mlflow[0].fMom[curind + 2 * total_num] = 0.0f;
			mlflow[0].fMom[curind + 3 * total_num] = 0.0f;
			phin = 0.0f;
		}
		else if ((flagsn & TYPE_SU) == TYPE_I && (phin < 0.0f || phin>1.0f)) {
			phin = 0.5f; // cell should be interface, but phi was invalid
		}
		else if ((flagsn & TYPE_SU) == TYPE_F) {
			phin = 1.0f;
		}
		mlflow[0].phi[curind] = phin;
		mlflow[0].mass[curind] = phin * mlflow[0].fMom[curind + 0 * total_num];
		mlflow[0].massex[curind] = 0.0f; // reset excess mass
		mlflow[0].flag[curind] = (MLLATTICENODE_SURFACE_FLAG)flagsn;

		// deal with the high order moment
		for (int i = 0; i < 27; i++)
		{
			feq[i] += w3d_gpu[i];
		}
		float invRho = 1.0 / mlflow[0].fMom[curind + 0 * total_num];
		float pixx = ((feq[1] + feq[2] + feq[7] + feq[8] + feq[9] + feq[10] + feq[13] + feq[14] + feq[15] + feq[16] + feq[19] + feq[20] + feq[21] + feq[22] + feq[23] + feq[24] + feq[25] + feq[26]));
		float pixy = (((feq[7] + feq[8] + feq[19] + feq[20] + feq[21] + feq[22]) - (feq[13] + feq[14] + feq[23] + feq[24] + feq[25] + feq[26])));
		float pixz = (((feq[9] + feq[10] + feq[19] + feq[20] + feq[23] + feq[24]) - (feq[15] + feq[16] + feq[21] + feq[22] + feq[25] + feq[26])));
		float piyy = ((feq[3] + feq[4] + feq[7] + feq[8] + feq[11] + feq[12] + feq[13] + feq[14] + feq[17] + feq[18] + feq[19] + feq[20] + feq[21] + feq[22] + feq[23] + feq[24] + feq[25] + feq[26]));
		float piyz = (((feq[11] + feq[12] + feq[19] + feq[20] + feq[25] + feq[26]) - (feq[17] + feq[18] + feq[21] + feq[22] + feq[23] + feq[24])));
		float pizz = ((feq[5] + feq[6] + feq[9] + feq[10] + feq[11] + feq[12] + feq[15] + feq[16] + feq[17] + feq[18] + feq[19] + feq[20] + feq[21] + feq[22] + feq[23] + feq[24] + feq[25] + feq[26]));

		pixx = pixx * invRho - cs2;
		pixy = pixy * invRho;
		pixz = pixz * invRho;
		piyy = piyy * invRho - cs2;
		piyz = piyz * invRho;
		pizz = pizz * invRho - cs2;
		
		mlflow[0].fMomPost[curind + 0 * total_num] = mlflow[0].fMom[curind + 0 * total_num];
		mlflow[0].fMomPost[curind + 1 * total_num] = mlflow[0].fMom[curind + 1 * total_num];
		mlflow[0].fMomPost[curind + 2 * total_num] = mlflow[0].fMom[curind + 2 * total_num];
		mlflow[0].fMomPost[curind + 3 * total_num] = mlflow[0].fMom[curind + 3 * total_num];
		mlflow[0].fMomPost[curind + 4 * total_num] = mlflow[0].fMom[curind + 4 * total_num] = pixx;
		mlflow[0].fMomPost[curind + 5 * total_num] = mlflow[0].fMom[curind + 5 * total_num] = pixy;
		mlflow[0].fMomPost[curind + 6 * total_num] = mlflow[0].fMom[curind + 6 * total_num] = pixz;
		mlflow[0].fMomPost[curind + 7 * total_num] = mlflow[0].fMom[curind + 7 * total_num] = piyy;
		mlflow[0].fMomPost[curind + 8 * total_num] = mlflow[0].fMom[curind + 8 * total_num] = piyz;
		mlflow[0].fMomPost[curind + 9 * total_num] = mlflow[0].fMom[curind + 9 * total_num] = pizz;

		for (int i = 0; i < 7; i++)
		{
			mlflow[0].gMom[curind + i * total_num] = geq[i];
			mlflow[0].gMomPost[curind + i * total_num] = geq[i];
		}
	}

}

__global__ void surface_1(
	mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num, int total_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_num + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		const unsigned char flagsn_sus = mlflow[0].flag[curind] & (TYPE_SU | TYPE_S); // extract SURFACE flags
		if (flagsn_sus == TYPE_IF)
		{
			for (int i = 1; i < 27; i++)
			{
				int dx = int(ex3d_gpu[i]);
				int dy = int(ey3d_gpu[i]);
				int dz = int(ez3d_gpu[i]);
				int x1 = x - dx;
				int y1 = y - dy;
				int z1 = z - dz;
				int ind_back = z1 * sample_num + y1 * sample_x + x1;
				const unsigned char flagsji = mlflow[0].flag[ind_back];
				const unsigned char flagsji_su = flagsji & (TYPE_SU | TYPE_S); // extract SURFACE flags
				const unsigned char flagsji_r = flagsji & ~TYPE_SU; // extract all non-SURFACE flags
				if (flagsji_su == TYPE_IG) mlflow[0].flag[ind_back] = (MLLATTICENODE_SURFACE_FLAG)(flagsji_r | TYPE_I); // prevent interface neighbor cells from becoming gas
				else if (flagsji_su == TYPE_G) mlflow[0].flag[ind_back] = (MLLATTICENODE_SURFACE_FLAG)(flagsji_r | TYPE_GI); // neighbor cell was gas and must change to interface
			}
		}
	}
}

__global__ void surface_2(
	mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num, int total_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	mrUtilFuncGpu3D mrutilfunc;
	int curind = z * sample_num + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		const unsigned char flagsn_sus = mlflow[0].flag[curind] & (TYPE_SU | TYPE_S); // extract SURFACE flags
		if (flagsn_sus == TYPE_GI) { // initialize the fi of gas cells that should become interface
			float rhon, uxn, uyn, uzn; // average over all fluid/interface neighbors
			float rhot = 0.0f, uxt = 0.0f, uyt = 0.0f, uzt = 0.0f, counter = 0.0f; // average over all fluid/interface neighbors
			float rho_gt = 0.f, c_k = 0.f;
			
			for (int i = 1; i < 27; i++)
			{
				int dx = int(ex3d_gpu[i]);
				int dy = int(ey3d_gpu[i]);
				int dz = int(ez3d_gpu[i]);
				int x1 = x - dx;
				int y1 = y - dy;
				int z1 = z - dz;
				int ind_back = z1 * sample_num + y1 * sample_x + x1;
				const unsigned char flagsji_sus = mlflow[0].flag[ind_back] & (TYPE_SU | TYPE_S); // extract SURFACE flags
				if (flagsji_sus == TYPE_F || flagsji_sus == TYPE_I || flagsji_sus == TYPE_IF) { // fluid or interface or (interface->fluid) neighbor
					counter += 1.0f;
					rhot += mlflow[0].fMomPost[ind_back + 0 * total_num];
					uxt += mlflow[0].fMomPost[ind_back + 1 * total_num];
					uyt += mlflow[0].fMomPost[ind_back + 2 * total_num];
					uzt += mlflow[0].fMomPost[ind_back + 3 * total_num];
					if (i < 7)
					{
						rho_gt += mlflow[0].c_value[ind_back];
						c_k += 1.0f;
					}
				}
			}
			rhon = counter > 0.0f ? rhot / counter : 1.0f;
			uxn = counter > 0.0f ? uxt / counter : 0.0f;
			uyn = counter > 0.0f ? uyt / counter : 0.0f;
			uzn = counter > 0.0f ? uzt / counter : 0.0f;

			rho_gt = c_k > 0.0f ? rho_gt / c_k : 0.0f;
			
			float feq[27];
			mrutilfunc.calculate_f_eq(rhon, uxn, uyn, uzn, feq); // calculate equilibrium DDFs
			for (int i = 0; i < 27; i++)
			{
				feq[i] += w3d_gpu[i];
			}
			float invRho = 1.0 / rhon;
			float pixx = ((feq[1] + feq[2] + feq[7] + feq[8] + feq[9] + feq[10] + feq[13] + feq[14] + feq[15] + feq[16] + feq[19] + feq[20] + feq[21] + feq[22] + feq[23] + feq[24] + feq[25] + feq[26]));
			float pixy = (((feq[7] + feq[8] + feq[19] + feq[20] + feq[21] + feq[22]) - (feq[13] + feq[14] + feq[23] + feq[24] + feq[25] + feq[26])));
			float pixz = (((feq[9] + feq[10] + feq[19] + feq[20] + feq[23] + feq[24]) - (feq[15] + feq[16] + feq[21] + feq[22] + feq[25] + feq[26])));
			float piyy = ((feq[3] + feq[4] + feq[7] + feq[8] + feq[11] + feq[12] + feq[13] + feq[14] + feq[17] + feq[18] + feq[19] + feq[20] + feq[21] + feq[22] + feq[23] + feq[24] + feq[25] + feq[26]));
			float piyz = (((feq[11] + feq[12] + feq[19] + feq[20] + feq[25] + feq[26]) - (feq[17] + feq[18] + feq[21] + feq[22] + feq[23] + feq[24])));
			float pizz = ((feq[5] + feq[6] + feq[9] + feq[10] + feq[11] + feq[12] + feq[15] + feq[16] + feq[17] + feq[18] + feq[19] + feq[20] + feq[21] + feq[22] + feq[23] + feq[24] + feq[25] + feq[26]));

			pixx = pixx * invRho - cs2;
			pixy = pixy * invRho;
			pixz = pixz * invRho;
			piyy = piyy * invRho - cs2;
			piyz = piyz * invRho;
			pizz = pizz * invRho - cs2;

			mlflow[0].fMomPost[curind + 0 * total_num] = rhon;
			mlflow[0].fMomPost[curind + 1 * total_num] = uxn;
			mlflow[0].fMomPost[curind + 2 * total_num] = uyn;
			mlflow[0].fMomPost[curind + 3 * total_num] = uzn;
			mlflow[0].fMomPost[curind + 4 * total_num] = pixx;
			mlflow[0].fMomPost[curind + 5 * total_num] = pixy;
			mlflow[0].fMomPost[curind + 6 * total_num] = pixz;
			mlflow[0].fMomPost[curind + 7 * total_num] = piyy;
			mlflow[0].fMomPost[curind + 8 * total_num] = piyz;
			mlflow[0].fMomPost[curind + 9 * total_num] = pizz;

			//recontruction for g
			mlflow[0].c_value[curind] = rho_gt;
			float geq[7];
			mrutilfunc.calculate_g_eq(rho_gt, uxn, uyn, uzn, geq); 
			for (int i = 0; i < 7; i++)
				mlflow[0].gMom[curind + i * total_num] = geq[i];

		}
		else if (flagsn_sus == TYPE_IG) { // flag interface->gas is set
			for (int i = 1; i < 27; i++)
			{
				int dx = int(ex3d_gpu[i]);
				int dy = int(ey3d_gpu[i]);
				int dz = int(ez3d_gpu[i]);
				int x1 = x - dx;
				int y1 = y - dy;
				int z1 = z - dz;
				int ind_back = z1 * sample_num + y1 * sample_x + x1;

				const unsigned char flagsji = mlflow[0].flag[ind_back];
				const unsigned char flagsji_su = flagsji & (TYPE_SU | TYPE_S); // extract SURFACE flags
				const unsigned char flagsji_r = flagsji & (~TYPE_SU); // extract all non-SURFACE flags

				if (flagsji_su == TYPE_F || flagsji_su == TYPE_IF) {
					if (mlflow[0].islet[ind_back] == 0)
					{
						mlflow[0].flag[ind_back] = (MLLATTICENODE_SURFACE_FLAG)(flagsji_r | TYPE_I); // prevent fluid or interface neighbors that turn to fluid from being/becoming fluid
						
						// if the node is changed to interface, we need to detect the merge.
						mlflow[0].merge_detector[ind_back] = 1;
					}
					else
					{
						mlflow[0].flag[curind] = (MLLATTICENODE_SURFACE_FLAG)(flagsji_r | TYPE_I);
					}
				}

			}

		}
	}
}

__global__ void surface_3(
	mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num, int total_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	mrUtilFuncGpu3D mrutilfunc;
	int curind = z * sample_num + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		const unsigned char flagsn_sus = mlflow[0].flag[curind] & (TYPE_SU | TYPE_S); // extract SURFACE flags
		if (flagsn_sus & TYPE_S) return;
		if (mlflow[0].islet[curind] == 1) 
		{	
			// use previous tag to record for the bubble volume change computation
			mlflow[0].previous_tag[curind] = mlflow[0].tag_matrix[curind];
			mlflow[0].tag_matrix[curind] = -1;
			return;
		}
		const float rhon = mlflow[0].fMomPost[curind + 0 * total_num]; // density of cell n
		float massn = mlflow[0].mass[curind]; // mass of cell n
		float massexn = 0.0f; // excess mass of cell n
		float phin = 0.0f;
		if (flagsn_sus == TYPE_F) { // regular fluid cell
			massexn = massn - rhon; // dump mass-rho difference into excess mass
			massn = rhon; // fluid cell mass has to equal rho
			phin = 1.0f;
			// use previous tag to record for the bubble volume change computation
			mlflow[0].previous_tag[curind] = mlflow[0].tag_matrix[curind];
			mlflow[0].tag_matrix[curind] = -1;
		}
		else if (flagsn_sus == TYPE_I) { // regular interface cell
			massexn = massn > rhon ? massn - rhon : massn < 0.0f ? massn : 0.0f; // allow interface cells with mass>rho or mass<0
			massn = clamp(massn, 0.0f, rhon);
			phin = mrutilfunc.calculate_phi(rhon, massn, TYPE_I); // calculate fill level for next step (only necessary for interface cells)
		}
		else if (flagsn_sus == TYPE_G) { // regular gas cell
			massexn = massn; // dump remaining mass into excess mass
			massn = 0.0f;
			phin = 0.0f;
		}
		else if (flagsn_sus == TYPE_IF) { // flag interface->fluid is set
			mlflow[0].flag[curind] = (MLLATTICENODE_SURFACE_FLAG)((mlflow[0].flag[curind] & ~TYPE_SU) | TYPE_F); // cell becomes fluid
			// if the node is changed to fluid, we need to report the split
			report_split(mlflow, x, y, z, sample_x, sample_y, sample_z); // report interface->fluid conversion
			massexn = massn - rhon; // dump mass-rho difference into excess mass		
			massn = rhon; // fluid cell mass has to equal rho
			phin = 1.0f; // set phi[n] to 1.0f for fluid cells
		}
		else if (flagsn_sus == TYPE_IG) { // flag interface->gas is set
			mlflow[0].flag[curind] = (MLLATTICENODE_SURFACE_FLAG)((mlflow[0].flag[curind] & ~TYPE_SU) | TYPE_G); // cell becomes gas
			massexn = massn; // dump remaining mass into excess mass
			massn = 0.0f; // gas mass has to be zero
			phin = 0.0f; // set phi[n] to 0.0f for gas cells
		}
		else if (flagsn_sus == TYPE_GI) { // flag gas->interface is set
			mlflow[0].flag[curind] = (MLLATTICENODE_SURFACE_FLAG)((mlflow[0].flag[curind] & ~TYPE_SU) | TYPE_I); // cell becomes interface
			massexn = massn > rhon ? massn - rhon : massn < 0.0f ? massn : 0.0f; // allow interface cells with mass>rho or mass<0
			massn = clamp(massn, 0.0f, rhon);
			phin = mrutilfunc.calculate_phi(rhon, massn, TYPE_I); // calculate fill level for next step (only necessary for interface cells)
		}
		int counter = 0; // count (fluid|interface) neighbors
		for (int i = 1; i < 27; i++)
		{
			int dx = int(ex3d_gpu[i]);
			int dy = int(ey3d_gpu[i]);
			int dz = int(ez3d_gpu[i]);
			int x1 = x - dx;
			int y1 = y - dy;
			int z1 = z - dz;

			int ind_back = z1 * sample_num + y1 * sample_x + x1;
			const unsigned char flagsji_su = mlflow[0].flag[ind_back] & (TYPE_SU | TYPE_S); // extract SURFACE flags
			counter += (int)(flagsji_su == TYPE_F || flagsji_su == TYPE_I || flagsji_su == TYPE_IF || flagsji_su == TYPE_GI); // avoid branching

		}
		massn += counter > 0 ? 0.0f : massexn; // if excess mass can't be distributed to neighboring interface or fluid cells, add it to local mass (ensure mass conservation)
		massexn = counter > 0 ? massexn / (float)counter : 0.0f; // divide excess mass up for all interface or fluid neighbors
		mlflow[0].mass[curind] = massn; // update mass
		mlflow[0].massex[curind] = massexn; // update excess mass
		mlflow[0].delta_phi[curind] = phin - mlflow[0].phi[curind];

		if ((mlflow[0].flag[curind] & (TYPE_SU | TYPE_S)) == TYPE_I)
		{
			float rhon_g = 0.f;
			for (int i = 0; i < 7; i++)
				rhon_g += mlflow[0].gMom[curind + i * total_num];
			mlflow[0].delta_g[curind] -= rhon_g * (mlflow[0].delta_phi[curind]);
		}

		mlflow[0].phi[curind] = phin; // update phi

	}
}

__global__ void stream_collide_bvh(
	mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num, int total_num, float N, float l0p, float roup, float labma,
	float u0p, int time)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_num + y * sample_x + x;
	mrUtilFuncGpu3D mrutilfunc;
	float Omega = 1 / ((1e-4) * 3.0f + 0.5f);
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		const unsigned char flagsn = mlflow[0].flag[curind]; // cache flags[n] for multiple readings
		const unsigned char flagsn_bo = flagsn & TYPE_BO, flagsn_su = flagsn & TYPE_SU; // extract boundary and surface flags
		if (flagsn_bo == TYPE_S || flagsn_su == TYPE_G) return;

		if (mlflow[0].islet[curind] == 1) 
			{
				mlflow[0].flag[curind] = TYPE_F;
				return;
			}

		float fhn[27]{};
		float fon[27]{};

		for (int i = 0; i < 27; i++)
		{

			int dx = int(ex3d_gpu[i]);
			int dy = int(ey3d_gpu[i]);
			int dz = int(ez3d_gpu[i]);
			int x1 = x - dx;
			int y1 = y - dy;
			int z1 = z - dz;
			int ind_back = z1 * sample_x * sample_y + y1 * sample_x + x1;

			if ((mlflow[0].flag[ind_back] & TYPE_BO) == TYPE_S)
			{
				float feq[27]{}; // f_equilibrium
				mrutilfunc.calculate_f_eq(mlflow[0].fMom[curind + total_num * 0], 0.f, 0.f, 0.f, feq);
				fhn[i] = feq[i];
			}
			else
			{
				float rhoVar = mlflow[0].fMom[ind_back + total_num * 0];
				float ux_t30 = mlflow[0].fMom[ind_back + total_num * 1];
				float uy_t30 = mlflow[0].fMom[ind_back + total_num * 2];
				float uz_t30 = mlflow[0].fMom[ind_back + total_num * 3];
				float pixx_t45 = mlflow[0].fMom[ind_back + total_num * 4];
				float pixy_t90 = mlflow[0].fMom[ind_back + total_num * 5];
				float pixz_t90 = mlflow[0].fMom[ind_back + total_num * 6];
				float piyy_t45 = mlflow[0].fMom[ind_back + total_num * 7];
				float piyz_t90 = mlflow[0].fMom[ind_back + total_num * 8];
				float pizz_t45 = mlflow[0].fMom[ind_back + total_num * 9];


				mrutilfunc.mlCalDistributionFourthOrderD3Q27AtIndex(
					rhoVar,
					ux_t30,
					uy_t30,
					uz_t30,
					pixx_t45,
					pixy_t90,
					pixz_t90,
					piyy_t45,
					piyz_t90,
					pizz_t45,
					i, fhn[i]
				);
				fhn[i] -= w3d_gpu[i];
			}

			float rhoVar = mlflow[0].fMom[curind + total_num * 0];
			float ux_t30 = mlflow[0].fMom[curind + total_num * 1];
			float uy_t30 = mlflow[0].fMom[curind + total_num * 2];
			float uz_t30 = mlflow[0].fMom[curind + total_num * 3];
			float pixx_t45 = mlflow[0].fMom[curind + total_num * 4];
			float pixy_t90 = mlflow[0].fMom[curind + total_num * 5];
			float pixz_t90 = mlflow[0].fMom[curind + total_num * 6];
			float piyy_t45 = mlflow[0].fMom[curind + total_num * 7];
			float piyz_t90 = mlflow[0].fMom[curind + total_num * 8];
			float pizz_t45 = mlflow[0].fMom[curind + total_num * 9];

			mrutilfunc.mlCalDistributionFourthOrderD3Q27AtIndex(
				rhoVar,
				ux_t30,
				uy_t30,
				uz_t30,
				pixx_t45,
				pixy_t90,
				pixz_t90,
				piyy_t45,
				piyz_t90,
				pizz_t45,
				i, fon[i]
			);
			fon[i] -= w3d_gpu[i];
		}

		float massn = mlflow[0].mass[curind];

		for (int i = 1; i < 27; i++)
		{
			int dx = int(ex3d_gpu[i]);
			int dy = int(ey3d_gpu[i]);
			int dz = int(ez3d_gpu[i]);
			int x1 = x - dx;
			int y1 = y - dy;
			int z1 = z - dz;

			int ind_back = z1 * sample_num + y1 * sample_x + x1;
			massn += mlflow[0].massex[ind_back]; // distribute excess mass from last step which is stored in neighbors
		}

		if (flagsn_su == TYPE_F) {
			for (int i = 1; i < 27; i++)
			{
				massn += fhn[i] - fon[i]; // neighbor is fluid or interface cell
			}
		}
		else if (flagsn_su == TYPE_I)
		{ // cell is interface
			float phij[27]; // cache fill level of neighbor lattice points
			for (int i = 1; i < 27; i++)
			{
				int dx = int(ex3d_gpu[i]);
				int dy = int(ey3d_gpu[i]);
				int dz = int(ez3d_gpu[i]);
				int x1 = x - dx;
				int y1 = y - dy;
				int z1 = z - dz;

				int ind_back = z1 * sample_num + y1 * sample_x + x1;
				phij[i] = mlflow[0].phi[ind_back]; // cache fill level of neighbor lattice points

				// deal with the boundary phi without any assumption about the wetting property
				if ((mlflow[0].flag[ind_back] & TYPE_BO) == TYPE_S)
				{
					for (int ijk = 0; ijk < 6; ijk++)
					{
						int x2 = x1 - int(ex3d_gpu[ijk + 1]);
						int y2 = y1 - int(ey3d_gpu[ijk + 1]);
						int z2 = z1 - int(ez3d_gpu[ijk + 1]);
						if (x2 >= 0 && x2 < sample_x && y2 >= 0 && y2 < sample_y && z2 >= 0 && z2 < sample_z)
						{
							int ind_k = z2 * sample_num + y2 * sample_x + x2;
							if ((mlflow[0].flag[ind_k] & TYPE_BO) != TYPE_S)
							{
								phij[i] = mlflow[0].phi[ind_k];
								break;
							}
						}
					}
				}
			}
			float rhon = 0.0f, uxn = 0.0f, uyn = 0.0f, uzn = 0.0f, rho_laplace = 0.0f; // no surface tension if rho_laplace is not overwritten later
			rhon = mlflow[0].fMom[curind + total_num * 0];
			uxn = mlflow[0].fMom[curind + total_num * 1];
			uyn = mlflow[0].fMom[curind + total_num * 2];
			uzn = mlflow[0].fMom[curind + total_num * 3];
			phij[0] = mrutilfunc.calculate_phi(rhon, massn, flagsn); // don't load phi[n] from memory, instead recalculate it with mass corrected by excess mass
			float curv = mrutilfunc.calculate_curvature(phij);
			
			float disjoint = mlflow[0].disjoin_force[curind];
			float disjoint_factor = 0.032;
			float rho_k = 1.f;
			// for bubble pressure
			if (mlflow[0].tag_matrix[curind] > 0)
			{
				rho_k = mlflow[0].bubble.rho[mlflow[0].tag_matrix[curind] - 1];
			}
			// for air layer surface tension  
			float def_6_sigma_k = def_6_sigma;
			if (mlflow[0].bubble.init_volume[mlflow[0].tag_matrix[curind] - 1] > 5000000.0)
			{
				def_6_sigma_k = 1e-6f;
			}
			// for small bubble surface tension
			if ((!(disjoint > 0)) && (def_6_sigma_k>1e-3) && (mlflow[0].bubble.volume[mlflow[0].tag_matrix[curind] - 1] < 64.0))
			{
				def_6_sigma_k = 2e-4;
			}
			// for bubble disappearance visual effect
			if (time > 320 * 180-5)
			{
				def_6_sigma_k = 0e-5;
				disjoint_factor = 0.0;
			}

			rho_laplace = def_6_sigma_k == 0.0f ? 0.0f : def_6_sigma_k * curv;
			
			float feg[27]; // reconstruct f from neighbor gas lattice points

			const float rho2tmp = 0.5f / rhon; // apply external volume force (Guo forcing, Krueger p.233f)
			float uxntmp = fma(mlflow[0].forcex[curind] * rhon, rho2tmp, uxn);// limit velocity (for stability purposes)
			float uyntmp = fma(mlflow[0].forcey[curind] * rhon, rho2tmp, uyn);// force term: F*dt/(2*rho)
			float uzntmp = fma(mlflow[0].forcez[curind] * rhon, rho2tmp, uzn);
			float3 u_2{ uxntmp,uyntmp,uzntmp };
			u_2 = normalizing_clamp(u_2, 0.4);
			uxntmp = u_2.x;
			uyntmp = u_2.y;
			uzntmp = u_2.z;

			mrutilfunc.calculate_f_eq(rho_k - rho_laplace - disjoint_factor * disjoint, uxntmp, uyntmp, uzntmp, feg); // calculate gas equilibrium DDFs with constant ambient pressure

			unsigned char flagsj_su[27]; // cache neighbor flags for multiple readings
			unsigned char flagsj_bo[27];
			for (int i = 1; i < 27; i++)
			{
				int dx = int(ex3d_gpu[i]);
				int dy = int(ey3d_gpu[i]);
				int dz = int(ez3d_gpu[i]);
				int x1 = x - dx;
				int y1 = y - dy;
				int z1 = z - dz;
				int ind_back = z1 * sample_num + y1 * sample_x + x1;
				flagsj_su[i] = mlflow[0].flag[ind_back] & TYPE_SU;
				flagsj_bo[i] = mlflow[0].flag[ind_back] & TYPE_BO;
			}
			for (int i = 1; i < 27; i++)
			{
				massn += flagsj_su[i] & (TYPE_F | TYPE_I) ? flagsj_su[i] == TYPE_F ? fhn[i] - fon[index3dInv_gpu[i]] : 0.5f * (phij[i] + phij[0]) * (fhn[i] - fon[index3dInv_gpu[i]]) : 0.0f; // neighbor is fluid or interface cell
			}
			for (int i = 1; i < 27; i++)
			{
				if (flagsj_su[i] == TYPE_G)
					fhn[i] = feg[index3dInv_gpu[i]] - fon[index3dInv_gpu[i]] + feg[i];
			}

		}
		mlflow[0].mass[curind] = massn;
		float pop[27]{};

		for (int i = 0; i < 27; i++)
		{
			pop[i] = fhn[i] + w3d_gpu[i];
		}

		float FX = mlflow[0].forcex[curind];
		float FY = mlflow[0].forcey[curind];
		float FZ = mlflow[0].forcez[curind];
		float rhoVar = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6] + pop[7] + pop[8] + pop[9] + pop[10] + pop[11] + pop[12] + pop[13] + pop[14] + pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26];
		
		//calculate streaming moments
		FX = FX * rhoVar;
		FY = FY * rhoVar;
		FZ = FZ * rhoVar;

		float invRho = 1 / rhoVar;
		float ux_t30 = ((pop[1] + pop[7] + pop[9] + pop[13] + pop[15] + pop[19] + pop[21] + pop[23] + pop[26]) - (pop[2] + pop[8] + pop[10] + pop[14] + pop[16] + pop[20] + pop[22] + pop[24] + pop[25]) + 0.5f * FX) * invRho;
		float uy_t30 = ((pop[3] + pop[7] + pop[11] + pop[14] + pop[17] + pop[19] + pop[21] + pop[24] + pop[25]) - (pop[4] + pop[8] + pop[12] + pop[13] + pop[18] + pop[20] + pop[22] + pop[23] + pop[26]) + 0.5f * FY) * invRho;
		float uz_t30 = ((pop[5] + pop[9] + pop[11] + pop[16] + pop[18] + pop[19] + pop[22] + pop[23] + pop[25]) - (pop[6] + pop[10] + pop[12] + pop[15] + pop[17] + pop[20] + pop[21] + pop[24] + pop[26]) + 0.5f * FZ) * invRho;

		float3 u_{ ux_t30,uy_t30,uz_t30 };
		u_ = normalizing_clamp(u_, 0.4);
		ux_t30 = u_.x;
		uy_t30 = u_.y;
		uz_t30 = u_.z;

		float pixx_t45 = ((pop[1] + pop[2] + pop[7] + pop[8] + pop[9] + pop[10] + pop[13] + pop[14] + pop[15] + pop[16] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]));
		float pixy_t90 = (((pop[7] + pop[8] + pop[19] + pop[20] + pop[21] + pop[22]) - (pop[13] + pop[14] + pop[23] + pop[24] + pop[25] + pop[26])));
		float pixz_t90 = (((pop[9] + pop[10] + pop[19] + pop[20] + pop[23] + pop[24]) - (pop[15] + pop[16] + pop[21] + pop[22] + pop[25] + pop[26])));
		float piyy_t45 = ((pop[3] + pop[4] + pop[7] + pop[8] + pop[11] + pop[12] + pop[13] + pop[14] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]));
		float piyz_t90 = (((pop[11] + pop[12] + pop[19] + pop[20] + pop[25] + pop[26]) - (pop[17] + pop[18] + pop[21] + pop[22] + pop[23] + pop[24])));
		float pizz_t45 = ((pop[5] + pop[6] + pop[9] + pop[10] + pop[11] + pop[12] + pop[15] + pop[16] + pop[17] + pop[18] + pop[19] + pop[20] + pop[21] + pop[22] + pop[23] + pop[24] + pop[25] + pop[26]));


		if (flagsn_su == TYPE_I)
		{
			bool TYPE_NO_F = true, TYPE_NO_G = true; // temporary flags for no fluid or gas neighbors
			for (int i = 1; i < 27; i++)
			{
				int dx = int(ex3d_gpu[i]);
				int dy = int(ey3d_gpu[i]);
				int dz = int(ez3d_gpu[i]);
				int x1 = x - dx;
				int y1 = y - dy;
				int z1 = z - dz;
				int ind_back = z1 * sample_num + y1 * sample_x + x1;
				const unsigned char flagsji_su = mlflow[0].flag[ind_back] & TYPE_SU; // extract SURFACE flags
				TYPE_NO_F = TYPE_NO_F && flagsji_su != TYPE_F;
				TYPE_NO_G = TYPE_NO_G && flagsji_su != TYPE_G;
			}
			float massn = mlflow[0].mass[curind];

			if (massn > rhoVar || TYPE_NO_G)
				mlflow[0].flag[curind] = (MLLATTICENODE_SURFACE_FLAG)((flagsn & ~TYPE_SU) | TYPE_IF); // set flag interface->fluid
			else if (massn < 0.0f || TYPE_NO_F)
			{
				mlflow[0].flag[curind] = (MLLATTICENODE_SURFACE_FLAG)((flagsn & ~TYPE_SU) | TYPE_IG); // set flag interface->gas
			}
		}


		for (int ij = -3;ij<3;ij++)
			for(int jk=-3;jk<3;jk++)
				for(int kh=-3;kh<3;kh++)
			{
				int x12 = x + jk;
				int y12 = y + ij;
				int z12 = z + kh;
				if (x12 >= 0 && x12 < sample_x && y12 >= 0 && y12 < sample_y && z12 >= 0 && z12 < sample_z)
				{
					int ind_back = z12 * sample_num + y12 * sample_x + x12;
					if (mlflow[0].tag_matrix[ind_back] > 0)
					{
						if (mlflow[0].bubble.volume[mlflow[0].tag_matrix[ind_back] - 1] < 5000000.0)
						{
							float xx = pixx_t45 * invRho - cs2; //-uxux;
							float yy = piyy_t45 * invRho - cs2;
							float zz = pizz_t45 * invRho - cs2;
							float xy = pixy_t90 * invRho;
							float xz = pixz_t90 * invRho;
							float yz = piyz_t90 * invRho;
							float fact2 = 4.0f;
							float vis = fact2 *sqrtf((xx*xx+2 * xy*xy+  2 * xz*xz+ yy*yy+ 2 *yz*yz +  zz*zz));
							Omega = 1 / ((vis+1e-4) * 3.0f + 0.5f); //1e-1
							break;
						}
					}
			}
		}

		mrutilfunc.mlGetPIAfterCollision(
			rhoVar,
			ux_t30,
			uy_t30,
			uz_t30,
			FX,
			FY,
			FZ,
			Omega,
			pixx_t45,
			pixy_t90,
			pixz_t90,
			piyy_t45,
			piyz_t90,
			pizz_t45
		);
		mlflow[0].fMomPost[curind + total_num * 0] = rhoVar;
		mlflow[0].fMomPost[curind + total_num * 1] = ux_t30 + FX * invRho / 2.0f;
		mlflow[0].fMomPost[curind + total_num * 2] = uy_t30 + FY * invRho / 2.0f;
		mlflow[0].fMomPost[curind + total_num * 3] = uz_t30 + FZ * invRho / 2.0f;
		mlflow[0].fMomPost[curind + total_num * 4] = (pixx_t45 * invRho - cs2);
		mlflow[0].fMomPost[curind + total_num * 5] = (pixy_t90 * invRho);
		mlflow[0].fMomPost[curind + total_num * 6] = (pixz_t90 * invRho);
		mlflow[0].fMomPost[curind + total_num * 7] = (piyy_t45 * invRho - cs2);
		mlflow[0].fMomPost[curind + total_num * 8] = (piyz_t90 * invRho);
		mlflow[0].fMomPost[curind + total_num * 9] = (pizz_t45 * invRho - cs2);
	}
}

__global__ void mrSolver3D_step2Kernel(
	mrFlow3D* mlflow, int sample_x, int sample_y, int sample_num)
{
	MomSwap(mlflow[0].fMom, mlflow[0].fMomPost);
}


__global__ void parse_label(mrFlow3D* mlflow, MLFluidParam3D* param, int sample_x, int sample_y, int sample_z)
{

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		if ((int)mlflow[0].label_matrix[curind] > 0)
		{
			int label = (int)mlflow[0].label_matrix[curind];
			atomicMax(&mlflow[0].bubble.label_num, (int)mlflow[0].label_matrix[curind]);
			atomicAdd(&mlflow[0].bubble.label_volume[label - 1], (double) (1. - mlflow[0].phi[curind]));
		}
	}
}


__global__ void InitTag(
	mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num)
{

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_y * sample_x + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		unsigned char flagsn = mlflow[0].flag[curind];
		const unsigned char flagsn_bo = flagsn & TYPE_BO; // extract boundary flags

		if (flagsn_bo == TYPE_S)
			mlflow[0].tag_matrix[curind] = -1;
		if (flagsn == TYPE_F)
			mlflow[0].tag_matrix[curind] = -1;
		mlflow[0].previous_tag[curind] = -1;
	}
}


__global__ void convertIntToUnsignedChar(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{

		const unsigned char flagsn_sus = mlflow[0].flag[curind] & (TYPE_SU | TYPE_S);
		const unsigned char flagsn_bo = mlflow[0].flag[curind] & TYPE_BO;
		if ((flagsn_sus == TYPE_G || flagsn_sus == TYPE_I) && (flagsn_bo != TYPE_S))
			mlflow[0].input_matrix[curind] = 255;
		else
			mlflow[0].input_matrix[curind] = 0;
	}

}



__global__ void create_bubble_label(mrFlow3D* mlflow)
{
	for (int i = 0; i < mlflow[0].bubble.label_num; i++)
	{
		mlflow[0].bubble.volume[i] = mlflow[0].bubble.label_volume[i];
		mlflow[0].bubble.init_volume[i] = mlflow[0].bubble.label_volume[i];
		mlflow[0].bubble.rho[i] = 1.0;
	}
	mlflow[0].bubble.bubble_count = mlflow[0].bubble.label_num;
}

__global__ void update_init_tag(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_y * sample_x + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		if ((int)mlflow[0].label_matrix[curind] > 0)
		{
			mlflow[0].tag_matrix[curind] = (int)mlflow[0].label_matrix[curind];
		}

	}

}

__global__ void print_label_num(mrFlow3D* d_mlflow)
{
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		printf("label count %d\n", d_mlflow[0].bubble.label_num);
	}
}
__global__ void print_bubble(mrFlow3D* d_mlflow)
{
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		for (int i = 0; i < d_mlflow[0].bubble.bubble_count; i++)
			printf("bubble %d volume %f init volume %f bubble rho %f\n", i, d_mlflow[0].bubble.volume[i], d_mlflow[0].bubble.init_volume[i], d_mlflow[0].bubble.rho[i]);
	}
}

__global__ void ResetLabelVolume(mrFlow3D* mlflow)
{
	for (int i = 0; i < mlflow[0].bubble.label_num; i++)
	{
		mlflow[0].bubble.label_volume[i] = 0.0;
		mlflow[0].bubble.label_init_volume[i] = 0.f;
	}
	mlflow[0].bubble.label_num = 0;
}


void InitBubble(mrFlow3D* mlflow, MLFluidParam3D* param)
{
	int sample_x = param->samples.x;
	int sample_y = param->samples.y;
	int sample_z = param->samples.z;
	int sample_num = sample_x * sample_y;
	int total_num = sample_num * sample_z;
	dim3 threads1(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
	dim3 grid1(
		ceil(REAL(sample_x) / threads1.x),
		ceil(REAL(sample_y) / threads1.y),
		ceil(REAL(sample_z) / threads1.z)
	);

	// set the boundary and fluid as -1
	InitTag << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	printf("InitBubble 2\n");
	
	// prepare for the input bool image
	convertIntToUnsignedChar << <grid1, threads1 >> > (mlflow, sample_x, sample_y, sample_z);
	printf("InitBubble 3\n");
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	
	// do labeling
	connectedComponentLabeling(mlflow, sample_x, sample_y, sample_z);
	printf("InitBubble 4\n");
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// parse the label
	parse_label << <grid1, threads1 >> > (mlflow, param, sample_x, sample_y, sample_z);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	print_label_num << <1, 1 >> > (mlflow);
	checkCudaErrors(hipDeviceSynchronize());
	// create the bubble with label
	create_bubble_label << <1, 1 >> > (mlflow);
	hipDeviceSynchronize();
	// update the tag matrix
	update_init_tag << <grid1, threads1 >> > (mlflow, sample_x, sample_y, sample_z, sample_num);
	hipDeviceSynchronize();
	ClearDectector(mlflow, param);
	print_bubble << <1, 1 >> > (mlflow);
	checkCudaErrors(hipDeviceSynchronize());
}




__device__ static double atomicExch(double *address, double val)
{
  return __longlong_as_double(atomicExch((unsigned long long int *) address, __double_as_longlong(val)));
}

// update the atmosphere for the open tank
__global__ void atmosphere_rho_update_kernel(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num, int total_num, float N, float l0p, float roup, float labma,
	float u0p, int time) {

		{
			int x = threadIdx.x + blockDim.x * blockIdx.x;
			int y = threadIdx.y + blockDim.y * blockIdx.y;
			int z = threadIdx.z + blockDim.z * blockIdx.z;
			int curind = z * sample_num + y * sample_x + x;
			if (
				(x >= 0 && x <= sample_x - 1) &&
				(y >= 0 && y <= sample_y - 1) &&
				(z >= 0 && z <= sample_z - 1)
				)
			{
				{
					if (x == sample_x - 2 || z > sample_z - 10)
					{
						if (mlflow[0].tag_matrix[curind] > 0 && mlflow[0].bubble.volume[mlflow[0].tag_matrix[curind] - 1]>1000000)
							float old = atomicExch(&mlflow[0].bubble.rho[mlflow[0].tag_matrix[curind] - 1], 1.0);
					}
				}
			}
		}
}


// update the atmosphere volume for the open tank
__global__ void atmosphere_volme_update_kernel(mrFlow3D* d_mlflow) {

	{
		if (threadIdx.x == 0 && blockIdx.x == 0) {
			for (int i = 0; i < d_mlflow[0].bubble.bubble_count; i++)
			{
				// update the volume for the atmosphere
				if (d_mlflow[0].bubble.rho[i] == 1.0)
					d_mlflow[0].bubble.init_volume[i] = d_mlflow[0].bubble.rho[i] * d_mlflow[0].bubble.volume[i];
			}
		}
	}
}

// update the volume of the bubble with delta_phi
__global__ void bubble_volume_update_kernel(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num, int total_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		const unsigned char flagsn_sus = mlflow[0].flag[curind] & (TYPE_SU | TYPE_S);
		if (mlflow[0].delta_phi[curind] != 0)
		{
			int tag = mlflow[0].tag_matrix[curind];
			if (tag <= 0)
			{
				tag = mlflow[0].previous_tag[curind];
				mlflow[0].previous_tag[curind] = -1;
			}
			atomicAdd(&mlflow[0].bubble.volume[tag - 1], (double)-mlflow[0].delta_phi[curind]);
			mlflow[0].delta_phi[curind] = 0;
		}
	}
}

void bubble_volume_update(mrFlow3D* mlflow, MLFluidParam3D* param)
{
	int sample_x = param->samples.x;
	int sample_y = param->samples.y;
	int sample_z = param->samples.z;

	int sample_num = sample_x * sample_y;
	int total_num = sample_num * sample_z;
	dim3 threads1(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
	dim3 grid1(
		ceil(REAL(sample_x) / threads1.x),
		ceil(REAL(sample_y) / threads1.y),
		ceil(REAL(sample_z) / threads1.z)
	);

	bubble_volume_update_kernel << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num, total_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}



// update the rho of the bubble
__global__ void bubble_rho_update_kernel(mrFlow3D* mlflow)
{
	for (int i = 0; i < mlflow[0].bubble.bubble_count; i++)
	{

		mlflow[0].bubble.rho[i] = mlflow[0].bubble.init_volume[i] / mlflow[0].bubble.volume[i];
	}
}


__global__ void MergeSplitDetectorKernel(mrFlow3D* mlflow, int* merge_flag, int* split_flag) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		*merge_flag = mlflow[0].merge_flag;
		*split_flag = mlflow[0].split_flag;
	}
}


//assign neighbor tag to the current node
__global__ void get_tag_kernel(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num)
{

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;

	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1) &&
		mlflow[0].merge_detector[curind] &&
		mlflow[0].tag_matrix[curind] == -1
		)
	{
		int thisCellID = mlflow[0].tag_matrix[curind];
		for (int i = 1; i < 27; i++)
		{

			int dx = int(ex3d_gpu[i]);
			int dy = int(ey3d_gpu[i]);
			int dz = int(ez3d_gpu[i]);
			int x1 = x - dx;
			int y1 = y - dy;
			int z1 = z - dz;

			int ind_back = z1 * sample_x * sample_y + y1 * sample_x + x1;
			if ((x1 >= 0 && x1 <= sample_x - 1) &&
				(y1 >= 0 && y1 <= sample_y - 1) &&
				(z1 >= 0 && z1 <= sample_z - 1)
				)
			{
				if (mlflow[0].tag_matrix[ind_back] > -1)
				{
					thisCellID = mlflow[0].tag_matrix[ind_back];
				}
			}
		}
		// use previous merge tag to avoid the confict
		mlflow[0].previous_merge_tag[curind] = (thisCellID > 0) ? thisCellID : -1;
	}
	else
	{
		mlflow[0].merge_detector[curind] = 0;
	}
}

// update the tag of the current node to avoid the confict
__global__ void assign_tag_kernel(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int sample_num, int total_num)
{

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		// use previous merge tag to avoid the confict
		if (mlflow[0].previous_merge_tag[curind] > 0 && mlflow[0].merge_detector[curind])

		{
			mlflow[0].tag_matrix[curind] = mlflow[0].previous_merge_tag[curind];
			mlflow[0].previous_merge_tag[curind] = -1;
		}
		else
		{
			mlflow[0].previous_merge_tag[curind] = -1;
		}
	}
}

// identify whether the merge happens or just the bubble moves
__global__ void recheck_merge_kernel(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int total_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;

	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1) &&
		mlflow[0].merge_detector[curind]
		)
	{
		mlflow[0].merge_detector[curind] = 0;
		int thisCellID = -1;
		for (int i = 1; i < 27; i++)
		{
			int dx = int(ex3d_gpu[i]);
			int dy = int(ey3d_gpu[i]);
			int dz = int(ez3d_gpu[i]);
			int x1 = x - dx;
			int y1 = y - dy;
			int z1 = z - dz;

			int ind_back = z1 * sample_x * sample_y + y1 * sample_x + x1;
			if ((x1 >= 0 && x1 <= sample_x - 1) &&
				(y1 >= 0 && y1 <= sample_y - 1) &&
				(z1 >= 0 && z1 <= sample_z - 1)
				)
			{
				if (mlflow[0].tag_matrix[ind_back] > -1)
				{
					if (thisCellID < 0)
					{
						thisCellID = mlflow[0].tag_matrix[ind_back];
					}
					else
					{
						if (thisCellID != mlflow[0].tag_matrix[ind_back])
						{
							atomicExch(&mlflow[0].merge_flag, 1);
						}

					}
				}
			}
		}

	}
}


__global__ void reset_label_volume(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	if (curind < mlflow[0].bubble.max_bubble_count)
	{
		mlflow[0].bubble.label_volume[curind] = 0;
		mlflow[0].bubble.label_init_volume[curind] = 0;
	}
}


__global__ void reduce_label_rho(mrFlow3D* mlflow, MLFluidParam3D* param, int sample_x, int sample_y, int sample_z)
{

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		if ((int)mlflow[0].label_matrix[curind] > 0)
		{
			int label = (int)mlflow[0].label_matrix[curind];
			int tag = mlflow[0].tag_matrix[curind] - 1;
			if (tag >= 0)
			{
				atomicMax(&mlflow[0].bubble.label_num, (int)mlflow[0].label_matrix[curind]);
				atomicAdd(&mlflow[0].bubble.label_volume[label - 1], (double)(1.f - mlflow[0].phi[curind]));
				atomicAdd(&mlflow[0].bubble.label_init_volume[label - 1], (double)(1.f - mlflow[0].phi[curind]) * mlflow[0].bubble.rho[tag]);
			}
			else
			{
				atomicMax(&mlflow[0].bubble.label_num, (int)mlflow[0].label_matrix[curind]);
				atomicAdd(&mlflow[0].bubble.label_volume[label - 1], (double)(1.f - mlflow[0].phi[curind]));
				atomicAdd(&mlflow[0].bubble.label_init_volume[label - 1], (double)(1.f - mlflow[0].phi[curind]) * 1.0);
			}
			mlflow[0].tag_matrix[curind] = label;
		}
		else
		{
			mlflow[0].tag_matrix[curind] = -1;
		}
	}
}

__global__ void bubble_list_swap(mrFlow3D* mlflow)
{
	MomSwap(mlflow[0].bubble.volume, mlflow[0].bubble.label_volume);
	MomSwap(mlflow[0].bubble.init_volume, mlflow[0].bubble.label_init_volume);
}


__global__ void num_rho_update_kernel(mrFlow3D* d_mlflow) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		d_mlflow[0].bubble.bubble_count = d_mlflow[0].bubble.label_num;
		d_mlflow[0].bubble.label_num = 0;
		for (int i = 0; i < d_mlflow[0].bubble.bubble_count; i++)
		{
			d_mlflow[0].bubble.rho[i] = d_mlflow[0].bubble.init_volume[i] / d_mlflow[0].bubble.volume[i];
		}
	}
}


void handle_merge_spilt(mrFlow3D* mlflow, MLFluidParam3D* param)
{
	int sample_x = param->samples.x;
	int sample_y = param->samples.y;
	int sample_z = param->samples.z;
	int sample_num = sample_x * sample_y;
	int total_num = sample_num * sample_z;
	dim3 threads1(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
	dim3 grid1(
		ceil(REAL(sample_x) / threads1.x),
		ceil(REAL(sample_y) / threads1.y),
		ceil(REAL(sample_z) / threads1.z)
	);

	// prepare for the input bool image
	convertIntToUnsignedChar << <grid1, threads1 >> > (mlflow, sample_x, sample_y, sample_z);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	connectedComponentLabeling(mlflow, sample_x, sample_y, sample_z);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// reduce the label volume with the results of CCL
	reset_label_volume << <grid1, threads1 >> > (mlflow, sample_x, sample_y, sample_z);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	reduce_label_rho << <grid1, threads1 >> > (mlflow, param, sample_x, sample_y, sample_z);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	bubble_list_swap << <1, 1 >> > (mlflow);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	num_rho_update_kernel << <1, 1 >> > (mlflow);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

}

void update_bubble(mrFlow3D* mlflow, MLFluidParam3D* param)
{
	// update the delta_phi
	bubble_volume_update(mlflow, param);
	// update the new rho with the updated volume
	bubble_rho_update_kernel << <1, 1 >> > (mlflow);
	hipDeviceSynchronize();

	// detect the merge and split
	int* d_merge_flag;
	int merge_flag;
	int* d_split_flag;
	int split_flag;
	hipMalloc(&d_merge_flag, sizeof(int));
	hipMalloc(&d_split_flag, sizeof(int));
	MergeSplitDetectorKernel << <1, 1 >> > (mlflow, d_merge_flag, d_split_flag);
	hipDeviceSynchronize();
	hipMemcpy(&merge_flag, d_merge_flag, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&split_flag, d_split_flag, sizeof(int), hipMemcpyDeviceToHost);

	if (merge_flag > 0 || split_flag > 0)
	{
		handle_merge_spilt(mlflow, param);
		// clear the merge/split detector
		ClearDectector(mlflow, param);
	}

}

__global__ void g_reconstruction(
	mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int total_num)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	mrUtilFuncGpu3D mrutilfunc;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		const unsigned char flagsn = mlflow[0].flag[curind]; // cache flags[n] for multiple readings
		const unsigned char flagsn_bo = flagsn & TYPE_BO, flagsn_su = flagsn & TYPE_SU; // extract boundary and surface flags
		if (flagsn_bo == TYPE_S || flagsn_su == TYPE_G) return; // cell processed here is fluid or interface

		// g temporary streaming
		float ghn[7];
		float gon[7];
		float g_eq_k[7];
		float rhon_g = 0.f, uxn_g = 0.f, uyn_g = 0.f, uzn_g = 0.f;

		for (int i = 0; i < 7; i++)
			rhon_g += mlflow[0].gMom[curind + i * total_num];

		mrutilfunc.calculate_g_eq(rhon_g, uxn_g, uyn_g, uzn_g, g_eq_k);

		// g_eq boundary condition
		for (int i = 0; i < 7; i++)
		{
			int dx = int(ex3d_gpu[i]);
			int dy = int(ey3d_gpu[i]);
			int dz = int(ez3d_gpu[i]);
			int x1 = x - dx;
			int y1 = y - dy;
			int z1 = z - dz;

			int ind_back = z1 * sample_x * sample_y + y1 * sample_x + x1;
			gon[i] = mlflow[0].gMom[curind + i * total_num];

			if ((mlflow[0].flag[ind_back] & TYPE_BO) == TYPE_S)
			{
				
				ghn[i] = g_eq_k[i];
			}
			else
			{
				ghn[i] = mlflow[0].gMom[ind_back + i * total_num];
			}

		}
		gon[0] = ghn[0];

		if (flagsn_su == TYPE_I)
		{ // cell is interface
			float  uxn = 0.0f, uyn = 0.0f, uzn = 0.0f, rho_laplace = 0.0f; // no surface tension if rho_laplace is not overwritten later
			uxn = mlflow[0].fMom[curind + total_num * 1];
			uyn = mlflow[0].fMom[curind + total_num * 2];
			uzn = mlflow[0].fMom[curind + total_num * 3];

			REAL rho_k = 1.f;
			if (mlflow[0].tag_matrix[curind] > 0)
			{
				rho_k = mlflow[0].bubble.rho[mlflow[0].tag_matrix[curind] - 1];
			}
			float in_rho = K_h / 4.f * rho_k;

			// henry's law
			float geg[7]{};
			mrutilfunc.calculate_g_eq(in_rho, uxn, uyn, uzn, geg);
			
			unsigned char flagsj_su[27]; // cache neighbor flags for multiple readings
			unsigned char flagsj_bo[27];
			for (int i = 1; i < 27; i++)
			{
				int dx = int(ex3d_gpu[i]);
				int dy = int(ey3d_gpu[i]);
				int dz = int(ez3d_gpu[i]);
				int x1 = x - dx;
				int y1 = y - dy;
				int z1 = z - dz;
				int ind_back = z1 * sample_x * sample_y + y1 * sample_x + x1;
				flagsj_su[i] = mlflow[0].flag[ind_back] & TYPE_SU;
				flagsj_bo[i] = mlflow[0].flag[ind_back] & TYPE_BO;
			}
			// bubble gas from interface
			float g_in = 0.f;
			for (int i = 1; i < 7; i++)
			{
				g_in += flagsj_su[i] == TYPE_F ? ghn[i] - gon[index3dInv_gpu[i]] : 0.f;
			}
			mlflow[0].delta_g[curind] += g_in;

			for (int i = 1; i < 7; i++)
			{
				if (flagsj_su[i] == TYPE_G)
					mlflow[0].gMom[curind + i * total_num] = geg[index3dInv_gpu[i]] - gon[index3dInv_gpu[i]] + geg[i];
			}
		}

	}
}


__global__ void g_stream_collide(
	mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int total_num, int time)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	mrUtilFuncGpu3D mrutilfunc;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		const unsigned char flagsn = mlflow[0].flag[curind]; // cache flags[n] for multiple readings
		const unsigned char flagsn_bo = flagsn & TYPE_BO, flagsn_su = flagsn & TYPE_SU; // extract boundary and surface flags
		if (flagsn_bo == TYPE_S || flagsn_su == TYPE_G) return;
		if (mlflow[0].islet[curind] == 1) return;	
		// inject the g's streaming
		REAL pop_g[7]{};
		
		float g_eq_k[7];
		float rhon_g = 0.f, uxn_g = 0.f, uyn_g = 0.f, uzn_g = 0.f;
		for (int i = 0; i < 7; i++)
			rhon_g += mlflow[0].gMom[curind + i * total_num];
		mrutilfunc.calculate_g_eq(rhon_g, uxn_g, uyn_g, uzn_g, g_eq_k);

		// g_eq boundary condition
		for (int i = 0; i < 7; i++)
		{
			int dx = int(ex3d_gpu[i]);
			int dy = int(ey3d_gpu[i]);
			int dz = int(ez3d_gpu[i]);
			int x1 = x - dx;
			int y1 = y - dy;
			int z1 = z - dz;

			int ind_back = z1 * sample_x * sample_y + y1 * sample_x + x1;

			if ((mlflow[0].flag[ind_back] & TYPE_BO) == TYPE_S)
			{
				pop_g[i] = g_eq_k[i];
			}
			else
			{
				pop_g[i] = mlflow[0].gMom[ind_back + i * total_num];
			}
		}

		float FX = mlflow[0].forcex[curind];
		float FY = mlflow[0].forcey[curind];
		float FZ = mlflow[0].forcez[curind];

		float rhon = 0.f, uxn = 0.f, uyn = 0.f, uzn = 0.f;
		float fxn = FX, fyn = FY, fzn = FZ;


		rhon = mlflow[0].fMom[curind + total_num * 0];
		uxn = mlflow[0].fMom[curind + total_num * 1];
		uyn = mlflow[0].fMom[curind + total_num * 2];
		uzn = mlflow[0].fMom[curind + total_num * 3];

		// D3Q7 g
		float g_eq[7];

		float rhon_gt = 0.f;
		for (int i = 0; i < 7; i++)
			rhon_gt += pop_g[i];

		mrutilfunc.calculate_g_eq(rhon_gt, uxn, uyn, uzn, g_eq);
		mlflow[0].c_value[curind] = rhon_gt;

		// cmr
		float w = 1.0f / 0.53f;
		float src = 0.f;
		mrutilfunc.mlConvertCmrMoment_d3q7(uxn,uyn,uzn,pop_g);
		mrutilfunc.mlConvertCmrMoment_d3q7(uxn,uyn,uzn,g_eq);
		float pop_out[7];
		float s[7];
		s[0] = 1.0f;
		s[1] = s[2] = s[3] = 1.0f / (0.1 * 4  + 0.5);
		s[4] = 1.5f;
		s[5] = s[6] = 1.5f;
		for (int i = 0; i < 7; i++)
		{
			src = mlflow[0].src[curind];
			pop_out[i] = fma(1.0f - s[i], pop_g[i], fma(s[i], g_eq[i], src));
		}
		mrutilfunc.mlConvertCmrF_d3q7(uxn,uyn,uzn,pop_out);
		for (int i = 0; i < 7; i++)
			mlflow[0].gMomPost[curind + i * total_num] = pop_out[i];
	}
}


__global__ void bubble_volume_g_update_kernel(mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int total_num, int time)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;
	int curind = z * sample_x * sample_y + y * sample_x + x;
	if (
		(x >= 0 && x <= sample_x - 1) &&
		(y >= 0 && y <= sample_y - 1) &&
		(z >= 0 && z <= sample_z - 1)
		)
	{
		const unsigned char flagsn_sus = mlflow[0].flag[curind] & (TYPE_SU | TYPE_S);
		float factor;
		if (mlflow[0].delta_g[curind] != 0)
		{
			int tag = mlflow[0].tag_matrix[curind];
			factor = 1.f;
			if (mlflow[0].flag[curind] == TYPE_I)
			{
				atomicAdd(&mlflow[0].bubble.init_volume[tag - 1], (double)1.f / 4.f * factor * mlflow[0].delta_g[curind] * mlflow[0].phi[curind]);
				
			}
			mlflow[0].delta_g[curind] = 0;
		}
	}
}

__global__ void mrSolver3D_g_step2Kernel(
	mrFlow3D* mlflow, int sample_x, int sample_y, int sample_z, int total_num)
{
	MomSwap(mlflow[0].gMom, mlflow[0].gMomPost);
}


void mrInit3DGpu(mrFlow3D* mlflow, MLFluidParam3D* param)
{
	int sample_x = param->samples.x;
	int sample_y = param->samples.y;
	int sample_z = param->samples.z;
	int sample_num = sample_x * sample_y;
	int total_num = sample_num * sample_z;
	dim3 threads1(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
	dim3 grid1(
		ceil(REAL(sample_x) / threads1.x),
		ceil(REAL(sample_y) / threads1.y),
		ceil(REAL(sample_z) / threads1.z)
	);
	Init3D << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num, total_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	InitBubble(mlflow, param);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	ResetLabelVolume << <1, 1 >> > (mlflow);
	hipDeviceSynchronize();

}



void g_handle(mrFlow3D* mlflow, MLFluidParam3D* param, int time)
{
	int sample_x = param->samples.x;
	int sample_y = param->samples.y;
	int sample_z = param->samples.z;
	int total_num = sample_x * sample_y * sample_z;
	dim3 threads1(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
	dim3 grid1(
		ceil(REAL(sample_x) / threads1.x),
		ceil(REAL(sample_y) / threads1.y),
		ceil(REAL(sample_z) / threads1.z)
	);
	// g reconstruction
	g_reconstruction << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			total_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	g_stream_collide << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			total_num, time
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// updated the volume of the bubble caused by g
	bubble_volume_g_update_kernel << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			total_num, time
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	mrSolver3D_g_step2Kernel << <1, 1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			total_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	
	// update the new rho with the updated volume
	bubble_rho_update_kernel << <1, 1 >> > (mlflow);
	checkCudaErrors(hipDeviceSynchronize());
}


void mrSolver3DGpu(mrFlow3D* mlflow, MLFluidParam3D* param, float N, float l0p, float roup, float labma,
	float u0p, int time_step)
{

	int sample_x = param->samples.x;
	int sample_y = param->samples.y;
	int sample_z = param->samples.z;
	int sample_num = sample_x * sample_y;
	int total_num = sample_num * sample_z;
	dim3 threads1(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
	dim3 grid1(
		ceil(REAL(sample_x) / threads1.x),
		ceil(REAL(sample_y) / threads1.y),
		ceil(REAL(sample_z) / threads1.z)
	);
	// calculate the disjoint force
	calculate_disjoint << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			total_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// clear the inlet with some time steps
	if (time_step == 180 * 320 - 5)
	{
		clear_inlet << <grid1, threads1 >> >
			(
				mlflow,
				sample_x, sample_y, sample_z,
				total_num
				);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
	}
	// update the atmosphere for the open tank
	atmosphere_rho_update_kernel << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num, total_num, N, l0p, roup,
			labma, u0p, time_step
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// update the atmosphere volume for the open tank
	atmosphere_volme_update_kernel << <1, 1 >> > (mlflow);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// FS stream collide
	stream_collide_bvh << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num, total_num, N, l0p, roup,
			labma, u0p, time_step
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// clear the disjoint force
	ResetDisjoinForce << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	surface_1 << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num, total_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	surface_2 << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num, total_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	surface_3 << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num, total_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	mrSolver3D_step2Kernel << <1, 1 >> >
		(
			mlflow,
			sample_x, sample_y,
			sample_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}


void coupling(mrFlow3D* mlflow, MLFluidParam3D* param, float N, float l0p, float roup, float labma,
	float u0p, int time_step)
{
	int sample_x = param->samples.x;
	int sample_y = param->samples.y;
	int sample_z = param->samples.z;
	int t = time_step;
	int sample_num = sample_x * sample_y;
	int total_num = sample_num * sample_z;
	dim3 threads1(BLOCK_NX, BLOCK_NY, BLOCK_NZ);
	dim3 grid1(
		ceil(REAL(sample_x) / threads1.x),
		ceil(REAL(sample_y) / threads1.y),
		ceil(REAL(sample_z) / threads1.z)
	);

	// assign neighbor tag to the current node
	get_tag_kernel << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// update the tag of the current node to avoid the confict
	assign_tag_kernel << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			sample_num, total_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// identify whether the merge happens or just the bubble moves
	recheck_merge_kernel << <grid1, threads1 >> >
		(
			mlflow,
			sample_x, sample_y, sample_z,
			total_num
			);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// update the bubble
	update_bubble(mlflow, param);
	// handle the g update
	g_handle(mlflow, param, time_step);
	// print the bubble
	if (time_step % 320 == 0 && time_step > 0)
		print_bubble << <1, 1 >> > (mlflow);
	checkCudaErrors(hipDeviceSynchronize());
}




